#include "hip/hip_runtime.h"
#include<iostream>
#include"stdio.h"
#include<cmath>
#include<opencv2/imgproc/imgproc.hpp>
#include<opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

//Kernel de CUDA para la función de la suma laplaceana em 1 dimensión, recibe una matriz con los puntos de la imagen general, la matriz de resultado, w como el ancho de la imagen, h como el largo de la imagen
__global__ void laplaceadno1D(int *src, int *dst, int w, int h) {
	
    int i = blockIdx.x*blockDim.x + threadIdx.x; //el id del thread en el que estamos trabajando en el kernel
    int filtro[9] = {1 , 1, 1, 1, -8, 1, 1, 1, 1}; //filtro laplaceano para la suma de puntos

    //Checar que el thread este dentro de los puntos permitidos en lo alto para que no de segmentation fault
    if(i > 0 && h - 1> i)
    	for(int j = 1; j < w - 1; j++) { //for para iterar sobre lo ancho de la imagen y obtener los puntos conforme pase el ciclo
		//variable de suma que va guardar el resultado de la suma con el filtro laplaceano
		int suma = src[(i - 1) * w + j - 1] * filtro[0] + src[i * w + j - 1] * filtro[1] + src[(i + 1) * w + j - 1] * filtro[2];
		suma += src[(i - 1) * w + j] * filtro[3] + src[i * w + j] * filtro[4] + src[(i + 1) * w + j] * filtro[5]; 
		suma += src[(i - 1) * w + j + 1] * filtro[6] + src[i * w + j + 1] * filtro[7] + src[(i + 1) * w + j + 1] * filtro[8];	    
		dst[i * w + j] = suma; //guardar resultado en la matriz de resultados
    	}

}


//Kernel de CUDA para la función de la suma laplaceana em 2 dimensión, recibe una matriz con los puntos de la imagen general, la matriz de resultado, w como el ancho de la imagen, h como el largo de la imagen
__global__ void laplaceadno2D(int *src, int *dst, int w, int h) {

    int i = blockIdx.x*blockDim.x + threadIdx.x; //el id del thread en el que estamos trabajando en el kernel en la dimensión X
    int j = blockIdx.y*blockDim.y + threadIdx.y; //el id del thread en el que estamos trabajando en el kernel en la dimensión Y
    int filtro[9] = {1 , 1, 1, 1, -8, 1, 1, 1, 1};

    //Checar que el thread este dentro de los puntos permitidos en lo alto para que no de segmentation fault
    if(i > 0 && h - 1 > i)
	//Checar que el thread este dentro de los puntos permitidos en lo alto para que no de segmentation fault
        if(j > 0 && j < w - 1) {
                //variable de suma que va guardar el resultado de la suma con el filtro laplaceano
		int suma = src[(i - 1) * w + j - 1] * filtro[0] + src[i * w + j - 1] * filtro[1] + src[(i + 1) * w + j - 1] * filtro[2];
                suma += src[(i - 1) * w + j] * filtro[3] + src[i * w + j] * filtro[4] + src[(i + 1) * w + j] * filtro[5];
                suma += src[(i - 1) * w + j + 1] * filtro[6] + src[i * w + j + 1] * filtro[7] + src[(i + 1) * w + j + 1] * filtro[8];
                dst[i * w + j] = suma; //guardar resultado en la matriz de resultados

        }

}

int main( int argc, char** argv )  {

    Mat src, gray, dst;//Variable Mat que guarda el src, los puntos de la imagen original, gray para transformar a grises la imagen y dst como la imagen resultado de la suma laplaceana
    src = imread( "salon.jpg" );//Carga de la imagen
    GaussianBlur( src, src, Size(3,3), 0, 0, BORDER_DEFAULT ); /// Quitando el 'ruido' haciendo un filtro gausseano de OpenCV, esto para facilitar la detección de puntos
    cvtColor( src, gray, CV_RGB2GRAY );//conversión de la imagen a grises
    dst = gray.clone();//clonar puntos para dst para que tenga el tamañao de la imagen original
    int *srcA = (int *) malloc(src.rows * src.cols * sizeof(int)); //inicialización de matriz que va copiar los puntos de la imagen original para luego pasarlo a un arreglo de CUDA
    int *dstA = (int *) malloc(src.rows * src.cols * sizeof(int)); //Inicialización de la matriz que va contener los resultados de la suma de la imagen y va recibir la copia del arreglo de CUDA
    int *dev_src, *dev_dst; //inicialización de los arreglos que va cargar los puntos pero en los kernel de CUDA
    //inicialización de la memoria
    hipMalloc( (void**)&dev_src, src.rows * src.cols * sizeof(int) );
    hipMalloc( (void**)&dev_dst, src.rows * src.cols * sizeof(int) );

    //verificar que haya 3 argumentos a la hora de ejcutar el programa
    if (argc != 4){
	cout << "Se esperaba la opcion, el numero de bloques y threads, en ese orden\n";	
	exit(-1);
    }
 
    //Variables que toman los parametros de ejecución del programa
    int opcion = atoi(argv[1]);
    int blocks = atoi(argv[2]);    
    int threads = atoi(argv[3]);

    //Copia de los puntos de la imagen original al arreglo
    for(int i = 0; i < src.rows; i++) 
	for (int j = 0; j < src.cols; j++)
	    srcA[i * src.cols + j] = src.at<uchar>(i, j);

    //Copia de los puntos de la imagen que están en el arreglo al arreglo de CUDA
    hipMemcpy( dev_src, srcA, src.rows * src.cols * sizeof(int), hipMemcpyHostToDevice );

    if (opcion == 1) //Ejecución de la función de la suma en 1 dimensión
	laplaceadno1D<<<blocks, threads>>>(dev_src, dev_dst, src.cols, src.rows);
    else if(opcion == 2) //Ejecución de la función de la suma en 2 dimensiones
	laplaceadno2D<<<blocks, threads>>>(dev_src, dev_dst, src.cols, src.rows);    	

    //Copia del arreglo de resultado de la suma de CUDA al arreglo 
    hipMemcpy( dstA, dev_dst, src.rows * src.cols * sizeof(int), hipMemcpyDeviceToHost );

    //For que va copiar los resultados de los puntos del arreglo a una variable Mat para que sea
    for(int i = 0; i < src.rows; i++) {
        for (int j = 0; j < src.cols; j++){
	    if (dstA[i * src.cols + j] > 255)
		dst.at<uchar>(i, j) = 255; 
	    else if (0 > dstA[i * src.cols + j])
		dst.at<uchar>(i, j) = 0; 
	    else
		dst.at<uchar>(i, j) = dstA[i * src.cols + j]; 
	 }
   }

    imshow("Original", src);
    imshow( "Resultado", dst );

    waitKey(0);
    free(srcA);
    free(dstA);
    return 0;
}

